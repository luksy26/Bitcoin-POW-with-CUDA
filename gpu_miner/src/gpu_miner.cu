#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

// #define RUN_BONUS
#define MY_BLOCK_SIZE 256
#define MAX_TX_LENGTH 50

__constant__ int d_MAX_NONCE = MAX_NONCE;

BYTE **transactions = NULL;		// list of strings with all transactions
BYTE **h_d_hashed_transactions; // list (on host) of strings (on device) with all hashed transactions
BYTE **d_d_hashed_transactions; // list (on device) of strings (on device) with all hashed transactions

/*
	Computes all combined hashes on 'tree_level'
	The lowest level of the Merkle Tree is 0 (starting with the leaves)
*/
__global__ void merkleTree(BYTE **hashed_tx, int num_tx, int tree_level)
{
	// calculate corresponding index in hashed_tx
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// if higher level hash is accumulated at this index
	if (idx % (1 << (tree_level + 1)) == 0)
	{
		int pair_idx = idx + (1 << tree_level);

		// if a pair element exists to combine and rehash with
		if (pair_idx < num_tx)
		{
			BYTE hash1_concat_hash2[SHA256_HASH_SIZE * 2];

			// hash1
			d_strcpy((char *)hash1_concat_hash2, (char *)hashed_tx[idx]);

			// hash1hash2
			d_strcpy((char *)hash1_concat_hash2 + d_strlen((const char *)hashed_tx[idx]), (char *)hashed_tx[pair_idx]);

			// sha256(hash1hash2), placed where hash1 originally was
			apply_sha256(hash1_concat_hash2, d_strlen((const char *)hash1_concat_hash2), hashed_tx[idx], 1);
		}
	}
}

// Searches for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
__global__ void findNonce(BYTE *block_content, size_t current_length, uint64_t *nonce, BYTE *difficulty)
{
	// calculate corresponding nonce value to check in this thread
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint64_t nonce_to_check = idx + 1;

	if (nonce_to_check > d_MAX_NONCE)
		return;

	char nonce_string[NONCE_SIZE];
	int nonce_length = intToString(nonce_to_check, nonce_string);

	BYTE block_content_tmp[BLOCK_SIZE], block_hash[SHA256_HASH_SIZE];

	// block_hash to check
	d_strcpy((char *)block_content_tmp, (char *)block_content);
	d_strcpy((char *)(block_content_tmp + current_length), nonce_string);

	// Check if nonce was not already found
	if (*nonce == 0)
	{	
		// sha256(block_hash)
		apply_sha256(block_content_tmp, d_strlen((const char *)block_content_tmp), block_hash, 1);

		// if nonce was not already found and the block_content satisfies the difficulty
		if (*nonce == 0 && compare_hashes(block_hash, difficulty) <= 0)
			// atomically update nonce only if it was 0 (i.e. it was not found)
			atomicCAS((unsigned long long *)nonce, 0, nonce_to_check);
	}
}

int main(int argc, char **argv)
{
	BYTE top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE *d_block_content;
	BYTE *d_difficulty;
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000";
	uint64_t *nonce;
	size_t current_length;
	hipEvent_t start, stop;

	// Top hash
#ifdef RUN_BONUS
	// we will read all transactions from data/inputs.txt

	/*
		Example inputs.txt (the same example given in the code skel):

		FROM_Alice__TO_Bob__5_BTC
		FROM_Charlie__TO_David__9_BTC
		FROM_Erin__TO_Frank__1_BTC
		FROM_Alice__TO_Frank__3_BTC
	*/

	FILE *in = fopen("data/inputs.txt", "r");
	if (in == NULL)
	{
		printf("Error opening inputs.txt!\n");
		return 1;
	}

	int num_transactions = 0;

	char buffer[MAX_TX_LENGTH];
	while (fgets(buffer, sizeof(buffer), in) != NULL)
	{
		// Remove newline character
		if (buffer[strlen(buffer) - 1] == '\n')
			buffer[strlen(buffer) - 1] = '\0';

		// We don't care about empty lines
		if (strlen(buffer) == 0)
			continue;

		// Reallocate memory for transactions
		transactions = (BYTE **)realloc(transactions, (num_transactions + 1) * sizeof(BYTE *));
		if (transactions == NULL)
		{
			fprintf(stderr, "Memory allocation for transaction list failed.\n");
			return 1;
		}

		// Allocate memory for the current transaction
		transactions[num_transactions] = (BYTE *)malloc(strlen(buffer) + 1); // +1 for null terminator
		if (transactions[num_transactions] == NULL)
		{
			fprintf(stderr, "Memory allocation for transaction string failed.\n");
			return 1;
		}

		// Copy the transaction data into the allocated memory
		strcpy((char *)transactions[num_transactions], buffer);

		// Increment the number of transactions
		++num_transactions;
	}

	// Close the file
	fclose(in);

	// the list itself is on the host
	h_d_hashed_transactions = (BYTE **)malloc(num_transactions * sizeof(BYTE *));

	for (int i = 0; i < num_transactions; ++i)
	{
		// but the pointers inside the list point to memory on the device
		hipMalloc((void **)&h_d_hashed_transactions[i], SHA256_HASH_SIZE * sizeof(BYTE));

		BYTE hashed_transaction[SHA256_HASH_SIZE];

		// hash the transaction
		apply_sha256(transactions[i], strlen((const char *)transactions[i]), hashed_transaction, 1);

		// copy the data on device
		hipMemcpy(h_d_hashed_transactions[i], hashed_transaction, SHA256_HASH_SIZE * sizeof(BYTE), hipMemcpyHostToDevice);
	}
	// this list is on the device
	hipMalloc(&d_d_hashed_transactions, num_transactions * sizeof(BYTE *));

	// copy the device pointers from the host list
	hipMemcpy(d_d_hashed_transactions, h_d_hashed_transactions, num_transactions * sizeof(BYTE *), hipMemcpyHostToDevice);

	int blocks_no_merkle = (num_transactions + MY_BLOCK_SIZE - 1) / MY_BLOCK_SIZE;

	startTiming(&start, &stop);

	// go through all levels of the merkle tree 
	for (int level = 0; 1 << level < num_transactions; ++level)
	{	
		merkleTree<<<blocks_no_merkle, MY_BLOCK_SIZE>>>(d_d_hashed_transactions, num_transactions, level);
		hipDeviceSynchronize();
	}

	float seconds_merkle = stopTiming(&start, &stop);

	// copy the device list back to the host
	hipMemcpy(h_d_hashed_transactions, d_d_hashed_transactions, num_transactions * sizeof(BYTE *), hipMemcpyDeviceToHost);
	// top_hash was accumulated in the first position after combining and rehashing many partial hashes
	hipMemcpy(top_hash, h_d_hashed_transactions[0], SHA256_HASH_SIZE * sizeof(BYTE), hipMemcpyDeviceToHost);

	// Print the computed top_hash and the execution time
	FILE *out = fopen("data/outputs.csv", "a");
	if (out != NULL)
	{
		fprintf(out, "%s,%.2f\n", top_hash, seconds_merkle);
		fclose(out);
	}
	else
	{
		printf("Error opening outputs.csv!\n");
	}

	// Free allocated memory used to compute top_hash
	for (int i = 0; i < num_transactions; i++)
	{
		free(transactions[i]);
		hipFree(h_d_hashed_transactions[i]);
	}
	free(transactions);
	free(h_d_hashed_transactions);
	hipFree(d_d_hashed_transactions);
#else
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
		tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
		tx1234[SHA256_HASH_SIZE * 2];

	apply_sha256(tx1, strlen((const char *)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char *)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char *)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char *)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char *)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char *)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char *)tx34), top_hash, 1);
#endif

	// prev_block_hash + top_hash
	strcpy((char *)block_content, (const char *)prev_block_hash);
	strcat((char *)block_content, (const char *)top_hash);
	current_length = strlen((char *)block_content);

	hipMalloc((void **)&d_block_content, current_length * sizeof(BYTE));
	hipMemcpy(d_block_content, block_content, current_length * sizeof(BYTE), hipMemcpyHostToDevice);

	hipMalloc((void **)&d_difficulty, SHA256_HASH_SIZE * sizeof(BYTE));
	hipMemcpy(d_difficulty, DIFFICULTY, SHA256_HASH_SIZE * sizeof(BYTE), hipMemcpyHostToDevice);

	hipMallocManaged(&nonce, sizeof(uint64_t));

	*nonce = 0;

	int block_size = MY_BLOCK_SIZE;
	int blocks_no = (MAX_NONCE + block_size - 1) / block_size;

	startTiming(&start, &stop);

	findNonce<<<blocks_no, block_size>>>(d_block_content, current_length, nonce, d_difficulty);

	float seconds = stopTiming(&start, &stop);

	if (*nonce != 0)
	{
		char nonce_string[NONCE_SIZE];
		sprintf(nonce_string, "%lu", *nonce);
		strcat((char *)block_content, nonce_string);
		apply_sha256(block_content, strlen((const char *)block_content), block_hash, 1);
	}
	else
		printf("nonce not found\n");

	printResult(block_hash, *nonce, seconds);

	hipFree(d_block_content);
	hipFree(d_difficulty);
	hipFree(nonce);

	return 0;
}
